#include "hip/hip_runtime.h"
struct CudafyMapReduceCudafyStringKeyIntValue
{
	__device__  CudafyMapReduceCudafyStringKeyIntValue()
	{
	}
	int Key;
	int Value;
};


// Project1.CudafyMapReduce
__device__  CudafyMapReduceCudafyStringKeyIntValue _map(int input);
// Project1.CudafyMapReduce
__device__   CudafyMapReduceCudafyStringKeyIntValue* Map( InputStruct* input, int inputLen0);
// Project1.CudafyMapReduce
__device__   CudafyMapReduceCudafyStringKeyIntValue* Reduce( CudafyMapReduceCudafyStringKeyIntValue* intermediateValues, int intermediateValuesLen0);
// Project1.CudafyMapReduce
extern "C" __global__  void Execute( InputStruct* input, int inputLen0,  CudafyMapReduceCudafyStringKeyIntValue* output, int outputLen0);
// Project1.CudafyMapReduce
__device__   CudafyMapReduceCudafyStringKeyIntValue* GetUniqueArray( CudafyMapReduceCudafyStringKeyIntValue* arr, int arrLen0);
// Project1.CudafyMapReduce
__device__  int StrCmp(int str1, int str2);

// Project1.CudafyMapReduce
__device__  CudafyMapReduceCudafyStringKeyIntValue _map(int input)
{
	int x = threadIdx.x;
	CudafyMapReduceCudafyStringKeyIntValue result = CudafyMapReduceCudafyStringKeyIntValue();
	result.Key = input;
	result.Value = 1;
	return result;
}
// Project1.CudafyMapReduce
__device__   CudafyMapReduceCudafyStringKeyIntValue* Map( InputStruct* input, int inputLen0)
{
	int x = threadIdx.x;
	 CudafyMapReduceCudafyStringKeyIntValue* array = new CudafyMapReduceCudafyStringKeyIntValue[inputLen0];
	for (int i = 0; i < inputLen0; i++)
	{
		CudafyMapReduceCudafyStringKeyIntValue cudafyStringKeyIntValue = _map(input[(i)].Value);
		int key = cudafyStringKeyIntValue.Key;
		int value = cudafyStringKeyIntValue.Value;
		array[(i)].Key = key;
		array[(i)].Value = value;
	}
	return array;
}
// Project1.CudafyMapReduce
__device__   CudafyMapReduceCudafyStringKeyIntValue* Reduce( CudafyMapReduceCudafyStringKeyIntValue* intermediateValues, int intermediateValuesLen0)
{
	int x = threadIdx.x;
	for (int i = 0; i < intermediateValuesLen0 - 1; i++)
	{
		for (int j = i + 1; j < intermediateValuesLen0; j++)
		{
			int num = StrCmp(intermediateValues[(i)].Key, intermediateValues[(j)].Key);
			bool flag = num > 0;
			if (flag)
			{
				CudafyMapReduceCudafyStringKeyIntValue cudafyStringKeyIntValue = intermediateValues[(i)];
				intermediateValues[(i)] = intermediateValues[(j)];
				intermediateValues[(j)] = cudafyStringKeyIntValue;
			}
		}
	}
	return GetUniqueArray(intermediateValues, intermediateValuesLen0);
}
// Project1.CudafyMapReduce
extern "C" __global__  void Execute( InputStruct* input, int inputLen0,  CudafyMapReduceCudafyStringKeyIntValue* output, int outputLen0)
{
	int x = threadIdx.x;
	 CudafyMapReduceCudafyStringKeyIntValue* array = Reduce(Map(input, inputLen0));
	for (int i = 0; i < arrayLen0; i++)
	{
		output[(i)].Key = array[(i)].Key;
		output[(i)].Value = array[(i)].Value;
	}
}
// Project1.CudafyMapReduce
__device__   CudafyMapReduceCudafyStringKeyIntValue* GetUniqueArray( CudafyMapReduceCudafyStringKeyIntValue* arr, int arrLen0)
{
	int x = threadIdx.x;
	int num = 0;
	 bool* array = new bool[arrLen0];
	for (int i = 0; i < arrayLen0; i++)
	{
		array[(i)] = false;
	}
	for (int j = 0; j < arrLen0; j++)
	{
		bool flag = false;
		for (int k = 0; k < j; k++)
		{
			bool flag2 = arr[(j)].Key == arr[(k)].Key;
			if (flag2)
			{
				flag = true;
				break;
			}
		}
		bool flag3 = !flag;
		if (flag3)
		{
			array[(j)] = true;
			num++;
		}
	}
	 CudafyMapReduceCudafyStringKeyIntValue* array2 = new CudafyMapReduceCudafyStringKeyIntValue[num];
	int num2 = 0;
	for (int l = 0; l < arrayLen0; l++)
	{
		bool flag4 = array[(l)];
		if (flag4)
		{
			array2[(num2)] = arr[(l)];
			num2++;
		}
	}
	return array2;
}
// Project1.CudafyMapReduce
__device__  int StrCmp(int str1, int str2)
{
	int x = threadIdx.x;
	bool flag = str1 == str2;
	int result;
	if (flag)
	{
		result = 0;
	}
	else
	{
		bool flag2 = str1 > str2;
		if (flag2)
		{
			result = 1;
		}
		else
		{
			result = -1;
		}
	}
	return result;
}
